#include "hip/hip_runtime.h"
#include "la/la-gpu.h"
#include <cmath>
#include <cassert>
#include <hipblas.h>
#include <cblas.h>
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>

namespace la {

    namespace gpu {

        __global__ void print_vec(double const *p, int size)
        {
            for (int i = 0; i < size; ++i) {
                printf("%f ", p[i]);
            }
            printf("\n");
        }

        __global__ void print_mat(double const *p, int rows, int cols)
        {
            printf("%d %d\n", rows, cols);
        
            for (int i = 0; i < rows; ++i) {
                for (int j = 0; j < cols; ++j) {
                    printf("%f ", p[j * rows + i]);
                }
                printf("\n");
            }
        }

        device device::d = device();

        device::device()
        {
            hipblasCreate(&handle);
        }

        device::~device()
        {
            hipblasDestroy(handle);
            hipDeviceReset();
        }

        device& device::get_instance()
        {
            return d;
        }

        hipblasHandle_t& device::get_handle()
        {
            return get_instance().handle;
        }

        void imul(vector<double>& u, double d)
        {
            hipblasDscal(device::get_handle(), u.size(), &d, u.data(), 1);
        }

        void iadd(vector<double>& u, vector<double> const& v)
        {
            assert(u.size() == v.size());

            double alpha = 1;
            hipblasDaxpy(device::get_handle(), u.size(), &alpha, v.data(), 1, u.data(), 1);
        }

        void isub(vector<double>& u, vector<double> const& v)
        {
            assert(u.size() == v.size());

            double alpha = -1;
            hipblasDaxpy(device::get_handle(), u.size(), &alpha, v.data(), 1, u.data(), 1);
        }

        void imul(vector<double>& u, vector<double> const& v)
        {
            assert(u.size() == v.size());

            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(u.begin()), thrust::device_ptr<double const>(v.begin()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(u.end()), thrust::device_ptr<double const>(v.end()))),
                imul_op());
        }

        void idiv(vector<double>& u, vector<double> const& v)
        {
            assert(u.size() == v.size());

            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(u.begin()), thrust::device_ptr<double const>(v.begin()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(u.end()), thrust::device_ptr<double const>(v.end()))),
                idiv_op());
        }

        vector<double> add(
            vector<double> u,
            vector<double> const& v)
        {
            iadd(u, v);
            return u;
        }

        double norm(vector<double> const& v)
        {
            double result = 0;
            hipblasDnrm2(device::get_handle(), v.size(), v.data(), 1, &result);
            return result;
        }

        double dot(vector<double> const& u, vector<double> const& v)
        {
            assert(u.size() == v.size());

            double result = 0;
            hipblasDdot(device::get_handle(), u.size(), u.data(), 1, v.data(), 1, &result);
            return result;
        }

        vector<double> logistic(vector<double> const& v)
        {
            vector<double> result;
            result.resize(v.size());

            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(result.begin()), thrust::device_ptr<double const>(v.begin()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(result.end()), thrust::device_ptr<double const>(v.end()))),
                ilogistic_op());

            return result;
        }

        void iadd(matrix<double>& u, matrix<double> const& v)
        {
            assert(u.rows() == v.rows());
            assert(u.cols() == v.cols());

            double alpha = 1;
            hipblasDaxpy(device::get_handle(), u.rows() * u.cols(), &alpha, v.data(), 1, u.data(), 1);
        }

        void isub(matrix<double>& u, matrix<double> const& v)
        {
            assert(u.rows() == v.rows());
            assert(u.cols() == v.cols());

            double alpha = -1;
            hipblasDaxpy(device::get_handle(), u.rows() * u.cols(), &alpha, v.data(), 1, u.data(), 1);
        }

        vector<double> mult(
            matrix<double> const& u,
            vector<double> const& v)
        {
            vector<double> result;
            result.resize(u.rows());

            double alpha = 1;
            double beta = 1;
            hipblasDgemv(device::get_handle(), HIPBLAS_OP_N,
                u.rows(), u.cols(), &alpha, u.data(), u.rows(),
                v.data(), 1, &beta, result.data(), 1);

            return result;
        }

        vector<double> lmult(
            matrix<double> const& u,
            vector<double> const& v)
        {
            vector<double> result;
            result.resize(u.cols());
    
            double alpha = 1;
            double beta = 1;
            hipblasDgemv(device::get_handle(), HIPBLAS_OP_T,
                u.rows(), u.cols(), &alpha, u.data(), u.rows(),
                v.data(), 1, &beta, result.data(), 1);
    
            return result;
        }
 
        vector<double> tensor_prod(vector<double> const& a,
            vector<double> const& b)
        {
            vector<double> result;
            result.resize(a.size() * b.size());

            double alpha = 1;
            hipblasDger(device::get_handle(), b.size(), a.size(),
                &alpha, b.data(), 1, a.data(), 1,
                result.data(), b.size());

            return result;
        }

        matrix<double> outer_prod(vector<double> const& a,
            vector<double> const& b)
        {
            matrix<double> result;
            result.resize(a.size(), b.size());

            double alpha = 1;
            hipblasDger(device::get_handle(), a.size(), b.size(),
                &alpha, a.data(), 1, b.data(), 1,
                result.data(), a.size());

            return result;
        }
    }
}
