#include "hip/hip_runtime.h"
#include "la/la-gpu.h"
#include <cmath>
#include <cassert>
#include <hipblas.h>
#include <cblas.h>
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>

namespace la {

    namespace gpu {

        __global__ void print_vec(double const *p, int size)
        {
            for (int i = 0; i < size; ++i) {
                printf("%f ", p[i]);
            }
            printf("\n");
        }

        __global__ void print_mat(double const *p, int rows, int cols)
        {
            printf("%d %d\n", rows, cols);
        
            for (int i = 0; i < rows; ++i) {
                for (int j = 0; j < cols; ++j) {
                    printf("%f ", p[j * rows + i]);
                }
                printf("\n");
            }
        }

        device device::d = device();

        device::device()
        {
            hipblasCreate(&handle);
        }

        device::~device()
        {
            hipblasDestroy(handle);
            hipDeviceReset();
        }

        device& device::get_instance()
        {
            return d;
        }

        hipblasHandle_t& device::get_handle()
        {
            return get_instance().handle;
        }

        // vector operations

        void zero(vector_like<double>& v)
        {
            hipMemset(v.data(), 0, v.size() * sizeof(double));
        }

        void imul(vector_like<double>& u, double d)
        {
            hipblasDscal(device::get_handle(), u.size(), &d, u.data(), 1);
        }

        vector<double> mul(vector_like<double> const& u, double d)
        {
            vector<double> result { u };
            imul(result, d);
            return result;
        }

        void iadd(vector_like<double>& u, vector_like<double> const& v)
        {
            assert(u.size() == v.size());

            double alpha = 1;
            hipblasDaxpy(device::get_handle(), u.size(), &alpha, v.data(), 1, u.data(), 1);
        }

        vector<double> add(
            vector_like<double> const& u,
            vector_like<double> const& v)
        {
            vector<double> result { u };
            iadd(result, v);
            return result;
        }

        void isub(vector_like<double>& u, vector_like<double> const& v)
        {
            assert(u.size() == v.size());

            double alpha = -1;
            hipblasDaxpy(device::get_handle(), u.size(), &alpha, v.data(), 1, u.data(), 1);
        }

        void idiv(vector_like<double>& u, vector_like<double> const& v)
        {
            assert(u.size() == v.size());

            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(u.begin()), thrust::device_ptr<double const>(v.begin()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(u.end()), thrust::device_ptr<double const>(v.end()))),
                idiv_op());
        }

        void emul(vector_like<double>& z, vector_like<double> const& u,
            vector_like<double> const& v)
        {
            assert(u.size() == v.size() && z.size() == v.size());

            double alpha = 1;
            double beta = 1;
            hipblasDgbmv(device::get_handle(), HIPBLAS_OP_N, u.size(), u.size(), 0, 0,
                &alpha, u.data(), 1, v.data(), 1, &beta, z.data(), 1);
        }

        void iemul(vector_like<double>& u, vector_like<double> const& v)
        {
            emul(u, u, v);
        }

        vector<double> emul(
            vector_like<double> const& u,
            vector_like<double> const& v)
        {
            vector<double> result;
            result.resize(u.size());
            emul(result, u, v);
            return result;
        }

        double norm(vector_like<double> const& v)
        {
            double result = 0;
            hipblasDnrm2(device::get_handle(), v.size(), v.data(), 1, &result);
            return result;
        }

        double dot(vector_like<double> const& u, vector_like<double> const& v)
        {
            assert(u.size() == v.size());

            double result = 0;
            hipblasDdot(device::get_handle(), u.size(), u.data(), 1, v.data(), 1, &result);
            return result;
        }

        // matrix operations

        void zero(matrix_like<double>& m)
        {
            hipMemset(m.data(), 0, m.rows() * m.cols() * sizeof(double));
        }

        void iadd(matrix_like<double>& u, matrix_like<double> const& v)
        {
            assert(u.rows() == v.rows());
            assert(u.cols() == v.cols());

            double alpha = 1;
            hipblasDaxpy(device::get_handle(), u.rows() * u.cols(), &alpha, v.data(), 1, u.data(), 1);
        }

        void isub(matrix_like<double>& u, matrix_like<double> const& v)
        {
            assert(u.rows() == v.rows());
            assert(u.cols() == v.cols());

            double alpha = -1;
            hipblasDaxpy(device::get_handle(), u.rows() * u.cols(), &alpha, v.data(), 1, u.data(), 1);
        }

        void mul(vector_like<double>& u, matrix_like<double> const& a,
            vector_like<double> const& v)
        {
            assert(u.size() == a.rows() && a.cols() == v.size());

            double alpha = 1;
            double beta = 1;
            hipblasDgemv(device::get_handle(), HIPBLAS_OP_N,
                a.rows(), a.cols(), &alpha, a.data(), a.rows(),
                v.data(), 1, &beta, u.data(), 1);
        }

        vector<double> mul(
            matrix_like<double> const& a,
            vector_like<double> const& v)
        {
            vector<double> result;
            result.resize(a.rows());

            mul(result, a, v);

            return result;
        }

        vector<double> lmul(
            matrix_like<double> const& u,
            vector_like<double> const& v)
        {
            vector<double> result;
            result.resize(u.cols());
    
            double alpha = 1;
            double beta = 1;
            hipblasDgemv(device::get_handle(), HIPBLAS_OP_T,
                u.rows(), u.cols(), &alpha, u.data(), u.rows(),
                v.data(), 1, &beta, result.data(), 1);
    
            return result;
        }
 
        vector<double> tensor_prod(vector_like<double> const& a,
            vector_like<double> const& b)
        {
            vector<double> result;
            result.resize(a.size() * b.size());

            double alpha = 1;
            hipblasDger(device::get_handle(), b.size(), a.size(),
                &alpha, b.data(), 1, a.data(), 1,
                result.data(), b.size());

            return result;
        }

        matrix<double> outer_prod(vector_like<double> const& a,
            vector_like<double> const& b)
        {
            matrix<double> result;
            result.resize(a.size(), b.size());

            double alpha = 1;
            hipblasDger(device::get_handle(), a.size(), b.size(),
                &alpha, a.data(), 1, b.data(), 1,
                result.data(), a.size());

            return result;
        }
    }
}
